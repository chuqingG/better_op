// #include "../../tvm/src/runtime/cuda/cuda_device_api.cc"
// #include "../../tvm/src/runtime/cuda/cuda_module.cc"

#include <dlpack/dlpack.h>
#include <tvm/driver/driver_api.h>
#include <tvm/runtime/contrib/papi.h>
#include <tvm/runtime/module.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/profiling.h>
#include <tvm/runtime/registry.h>

#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <string>
#include <vector>
#include <memory>

#include "utils.h"

#pragma GCC diagnostic ignored "-Wformat="

void initArray(DLTensor* input, const size_t size) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dist(-1., 1.);

  for (size_t i = 0; i < size; ++i)
    static_cast<float*>(input->data)[i] = dist(gen);
}

constexpr int64_t seq_len = 100;
constexpr int64_t batch_size = 128;
constexpr int64_t input_size = 512;
constexpr int64_t hidden_size = 512;

// void lstm_cell() {
//   
//   This function works only if the .so built from te
//   
  // DLDevice dev{kDLCUDA, 0};
  // // DLDevice dev{kDLCPU, 0};
  // std::string lib_path = "lib/cell_b128_h128.so";

  // bool enabled = tvm::runtime::RuntimeEnabled("cuda");
  
  // const tvm::runtime::PackedFunc* graph_executor_create =
  //     tvm::runtime::Registry::Get("tvm.graph_executor.create");
  
  // auto lib = tvm::runtime::Module::LoadFromFile(lib_path);
  // tvm::runtime::PackedFunc f = lib.GetFunction("cell");
  // ICHECK(f != nullptr);

  // DLTensor* x;
  // DLTensor* h;
  // DLTensor* c;
  // DLTensor* w;
  // DLTensor* u;
  // DLTensor* ht;
  // DLTensor* ct;

  // int ndim = 2;
  // int dtype_code = kDLFloat;
  // int dtype_bits = 32;
  // int dtype_lanes = 1;
  // int device_type = kDLCPU;
  // int device_id = 0;

  // int64_t input_shape[2] = {batch_size, input_size};
  // DLTensor* x;
  // TVMArrayAlloc(input_shape, 2 /*ndim*/, dtype_code, dtype_bits, dtype_lanes,
  //               device_type, device_id, &x);
  // initArray(x, batch_size * input_size);

  // int64_t state_shape[2] = {batch_size, hidden_size};
  // DLTensor* h_prev;
  // TVMArrayAlloc(state_shape, 2 /*hdim*/, dtype_code, dtype_bits, dtype_lanes,
  //               device_type, device_id, &h_prev);
  // initArray(h_prev, batch_size * hidden_size);

  // DLTensor* c_prev;
  // TVMArrayAlloc(state_shape, 2 /*hdim*/, dtype_code, dtype_bits, dtype_lanes,
  //               device_type, device_id, &c_prev);
  // initArray(c_prev, batch_size * hidden_size);

  // int64_t output_shape[2] = {batch_size, hidden_size};
  // DLTensor* y;
  // TVMArrayAlloc(output_shape, 2 /*hdim*/, dtype_code, dtype_bits, dtype_lanes,
  //               device_type, device_id, &y);
  // initArray(y, batch_size * hidden_size);

  // set_input("input", x);
  // set_input("h_prev", h_prev);
  // set_input("c_prev", c_prev);

  // run();
  // get_output(0, y);

  // TVMArrayFree(x);
  // TVMArrayFree(h_prev);
  // TVMArrayFree(c_prev);
  // TVMArrayFree(y);
// }

// int main() {
//   lstm_cell();
//   return 0;
// }

template <typename T>
void lstm_cell(pybind11::array_t<T> x_pb,
                pybind11::array_t<T> h_pb,
                pybind11::array_t<T> c_pb,
                pybind11::array_t<T> w_pb,
                pybind11::array_t<T> u_pb,
                pybind11::array_t<T> ht_pb,
                pybind11::array_t<T> ct_pb){
  //This function only works for .so built from te

  //batchsize, hidden_size can be known from x.shape = [bs, hs]
  
  pybind11::buffer_info x_bf = x_pb.request();
  pybind11::buffer_info h_bf = h_pb.request();
  pybind11::buffer_info c_bf = c_pb.request();
  pybind11::buffer_info w_bf = w_pb.request();
  pybind11::buffer_info u_bf = u_pb.request();
  pybind11::buffer_info ht_bf = ht_pb.request();
  pybind11::buffer_info ct_bf = ct_pb.request();

  int64_t batch_size = x_bf.shape[0];
  int64_t hidden_size = x_bf.shape[1];

  DLDevice dev{kDLCUDA, 0};
  // DLDevice dev{kDLCPU, 0};
  // TODO: change to params here
  std::string lib_path = "lib/cell_b128_h128.so";

  bool enabled = tvm::runtime::RuntimeEnabled("cuda");
  
  const tvm::runtime::PackedFunc* graph_executor_create =
      tvm::runtime::Registry::Get("tvm.graph_executor.create");
  
  auto lib = tvm::runtime::Module::LoadFromFile(lib_path);
  tvm::runtime::PackedFunc f = lib.GetFunction("cell");
  ICHECK(f != nullptr);

  DLTensor* x;
  DLTensor* h;
  DLTensor* c;
  DLTensor* w;
  DLTensor* u;
  DLTensor* ht;
  DLTensor* ct;

  int ndim = 2;
  int dtype_code = kDLFloat;
  int dtype_bits = 32;
  int dtype_lanes = 1;
  int device_type = kDLCUDA;
  int device_id = 0;

  int64_t x_shape[2] = {batch_size, hidden_size};
  int64_t w_shape[2] = {4 * hidden_size, hidden_size};
  
  auto x_len = batch_size * hidden_size;
  auto w_len = 4 * hidden_size * hidden_size;

  std::cout << x_shape[0] << " " << x_shape[1] << std::endl;

  TVMArrayAlloc(x_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &x);
  TVMArrayAlloc(x_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &h);
  TVMArrayAlloc(x_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &c);
  TVMArrayAlloc(w_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &w);
  TVMArrayAlloc(w_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &u);
  TVMArrayAlloc(x_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &ht);
  TVMArrayAlloc(x_shape, ndim, dtype_code, dtype_bits, 
                dtype_lanes, device_type, device_id, &ct);

  T* p_x = reinterpret_cast<T*>(x_bf.ptr);
  T* p_h = reinterpret_cast<T*>(h_bf.ptr);
  T* p_c = reinterpret_cast<T*>(c_bf.ptr);
  T* p_w = reinterpret_cast<T*>(w_bf.ptr);
  T* p_u = reinterpret_cast<T*>(c_bf.ptr);
  T* p_ht = reinterpret_cast<T*>(ht_bf.ptr);
  T* p_ct = reinterpret_cast<T*>(ct_bf.ptr);

  hipMemcpy(static_cast<T*>(x->data), p_x, x_len * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(static_cast<T*>(h->data), p_h, x_len * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(static_cast<T*>(c->data), p_c, x_len * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(static_cast<T*>(w->data), p_w, w_len * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(static_cast<T*>(u->data), p_u, w_len * sizeof(T), hipMemcpyHostToDevice);

  f(x, h, c, w, u, ht, ct);
  hipMemcpy(p_ht, static_cast<T*>(ht->data), x_len * sizeof(T), hipMemcpyDeviceToHost);
  hipMemcpy(p_ct, static_cast<T*>(ct->data), x_len * sizeof(T), hipMemcpyDeviceToHost);
  
  TVMArrayFree(x);
  TVMArrayFree(h);
  TVMArrayFree(c);
  TVMArrayFree(w);
  TVMArrayFree(u);
  TVMArrayFree(ht);
  TVMArrayFree(ct);
  return;
}