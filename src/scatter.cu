#include <sstream>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
// #include <pybind11/eigen.h>
#include <hip/hip_runtime.h>
// #include "binding.cu"
#include "utils.h"
#include "tensor.h"
#include "kernel.h"
#pragma GCC diagnostic ignored "-Wformat="


template <typename T>
void scatter_nd(pybind11::array_t<T> output_pb, 
                pybind11::array_t<int64_t> indices_pb,
                pybind11::array_t<T> update_pb,
                pybind11::str log_path = "result.txt"){
    pybind11::buffer_info output = output_pb.request();
    pybind11::buffer_info indices = indices_pb.request();
    pybind11::buffer_info update = update_pb.request();

    auto idx_shape = indices.shape;
    auto idx_shape_size = indices.ndim;

    auto output_shape = output.shape;
    auto output_shape_size = output.ndim;

    int64_t each_idx_len = idx_shape[idx_shape_size - 1];
    int64_t idx_number = 1;
    for(int i = 0; i< idx_shape_size - 1; i++)
        idx_number *= idx_shape[i];

    int64_t each_update_size = 1;
    for(int64_t i = each_idx_len; i < output_shape_size; i++)
        each_update_size *= output_shape[i];

    int block = 512;
    int64_t n = each_update_size * idx_number;
    int64_t grid = (n + block - 1) / block;

    // init params on device
    int64_t* g_output_shape;
    cudaCheck(hipMalloc(&g_output_shape, output_shape_size * sizeof(int64_t)));
    cudaCheck(hipMemcpy(g_output_shape, vec2ptr(output_shape),
                output_shape_size * sizeof(int64_t),
                hipMemcpyHostToDevice));

    // create tensor
    T* p_output = (T*)(output.ptr);
    T* p_update = (T*)(update.ptr);
    int64_t* p_idx = (int64_t*)(indices.ptr);

    testtest();

    // std::cout << typeid(p_output).name() << std::endl;

    // Tensor<T> output_t(output_shape, p_output);
    // Tensor<T> update_t(update.shape, p_update);
    // Tensor<int64_t> idx_t(idx_shape, p_idx);

    // // open log_path
    // std::ofstream fs;
    // fs.open(log_path,std::ios::out|std::ios::app);

    // // bad scalability: {bs, dep, seq, hs}
    // std::string setting = "(dep, seq, bs, hs) = (" +
    //                         std::to_string(output_shape[1]) + ", " +
    //                         std::to_string(output_shape[2]) + ", " +
    //                         std::to_string(output_shape[0]) + ", " +
    //                         std::to_string(output_shape[3]) + ")";

    // timeKeep(1000, 10, setting, fs, 
    //         call_scatter(grid, block, 
    //                 update_t.data(), idx_t.data(), output_t.mutable_data(),
    //                 g_output_shape, idx_number, each_update_size, each_idx_len));
    
    // fs.close();

}


// void scatter_nd_noT(pybind11::array_t<float> output_pb, 
//                 pybind11::array_t<int64_t> indices_pb,
//                 pybind11::array_t<float> update_pb,
//                 pybind11::str log_path = "result.txt"){
//     pybind11::buffer_info output = output_pb.request();
//     pybind11::buffer_info indices = indices_pb.request();
//     pybind11::buffer_info update = update_pb.request();

//     auto idx_shape = indices.shape;
//     auto idx_shape_size = indices.ndim;

//     auto output_shape = output.shape;
//     auto output_shape_size = output.ndim;

//     int64_t each_idx_len = idx_shape[idx_shape_size - 1];
//     int64_t idx_number = 1;
//     for(int i = 0; i< idx_shape_size - 1; i++)
//         idx_number *= idx_shape[i];

//     int64_t each_update_size = 1;
//     for(int64_t i = each_idx_len; i < output_shape_size; i++)
//         each_update_size *= output_shape[i];

//     int block = 512;
//     int64_t n = each_update_size * idx_number;
//     int64_t grid = (n + block - 1) / block;

//     // init params on device
//     int64_t* g_output_shape;
//     cudaCheck(hipMalloc(&g_output_shape, output_shape_size * sizeof(int64_t)));
//     cudaCheck(hipMemcpy(g_output_shape, vec2ptr(output_shape),
//                 output_shape_size * sizeof(int64_t),
//                 hipMemcpyHostToDevice));

//     // create tensor
//     float* p_output = (float*)(output.ptr);
//     float* p_update = (float*)(update.ptr);
//     int64_t* p_idx = (int64_t*)(indices.ptr);

//     // int64_t output_shape_list[] = new int64_t(output.ndim);
//     // int64_t update_shape_list[] = new int64_t(update.ndim);
//     // int64_t indice_shape_list[] = new int64_t(indice.ndim);
//     // int64_t output_shape_list[] = vec2list<int64_t>(output_shape);
//     // int64_t update_shape_list[] = vec2list<int64_t>(update.shape);
//     // int64_t indice_shape_list[] = vec2list<int64_t>(idx_shape);

//     // std::cout << typeid(p_output).name() << std::endl;

//     Tensor<T> output_t(output_shape, p_output);
//     // Tensor<T> update_t(update.shape, p_update);
//     // Tensor<int64_t> idx_t(idx_shape, p_idx);

//     // // open log_path
//     // std::ofstream fs;
//     // fs.open(log_path,std::ios::out|std::ios::app);

//     // // bad scalability: {bs, dep, seq, hs}
//     // std::string setting = "(dep, seq, bs, hs) = (" +
//     //                         std::to_string(output_shape[1]) + ", " +
//     //                         std::to_string(output_shape[2]) + ", " +
//     //                         std::to_string(output_shape[0]) + ", " +
//     //                         std::to_string(output_shape[3]) + ")";

//     // timeKeep(1000, 10, setting, fs, 
//     //         call_scatter(grid, block, 
//     //                 update_t.data(), idx_t.data(), output_t.mutable_data(),
//     //                 g_output_shape, idx_number, each_update_size, each_idx_len));
    
//     // fs.close();

// }